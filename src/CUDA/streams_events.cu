#include <CUDA/include/streams_events.h>
#include <hip/hip_runtime.h>

hipStream_t d_Streams[GPU_MAX][CUDA_STREAM_MAX];
hipEvent_t d_Events[GPU_MAX][FRAME_COUNT][CUDA_EVENT_MAX];

/* Create the CUDA streams and events. */
void streams_events_init(uint8_t tid)
{

    for(uint8_t i = 0; i < CUDA_STREAM_MAX; ++i)
        CHECK(hipStreamCreateWithFlags(&d_Streams[tid][i], hipStreamNonBlocking));


    for(uint8_t i = 0; i < FRAME_COUNT; ++i)
    {
        for(uint8_t j = 0; j < CUDA_EVENT_MAX; ++j)
            CHECK(hipEventCreateWithFlags(&d_Events[tid][i][j], hipEventDisableTiming | hipEventBlockingSync));
    }
}

void streams_events_free(uint8_t tid)
{
    for(uint8_t i = 0; i < CUDA_STREAM_MAX; ++i)
        CHECK(hipStreamDestroy(d_Streams[tid][i]));

    for(uint8_t i = 0; i < FRAME_COUNT; ++i)
    {
        for(uint8_t j = 0; j < CUDA_EVENT_MAX; ++j)
            CHECK(hipEventDestroy(d_Events[tid][i][j]));
    }
}


hipError_t stream_wait_events(uint8_t tid, uint8_t frame_index, uint8_t sid, uint8_t eid_b, uint8_t eid_e)
{
    hipError_t err;
    hipError_t r = hipSuccess;
    for(uint8_t e = eid_b; e <= eid_e; ++e)
    {
        err = hipStreamWaitEvent(d_Streams[tid][sid], d_Events[tid][frame_index][e], 0);
        if(err != hipSuccess)
        {
             r = err;
             break;
        }
    }
    return r;
}


hipError_t streams_wait_event(uint8_t tid, uint8_t frame_index, uint8_t sid_b, uint8_t sid_e, uint8_t eid)
{
    hipError_t e;
    hipError_t r = hipSuccess;
    for(uint8_t s = sid_b; s <= sid_e; ++s)
    {
        e = hipStreamWaitEvent(d_Streams[tid][s], d_Events[tid][frame_index][eid], 0);
        if(e != hipSuccess)
        {
             r = e;
             break;
        }
    }
    return r;
}


hipError_t streams_signal_events(uint8_t tid, uint8_t frame_index, uint8_t sid_b, uint8_t sid_e)
{
    hipError_t e;
    hipError_t r = hipSuccess;
    for(uint8_t s = sid_b; s <= sid_e; ++s)
    {
        e = hipEventRecord(d_Events[tid][frame_index][s], d_Streams[tid][s]);
        if(e != hipSuccess)
        {
             r = e;
             break;
        }
    }
    return r;
}


hipError_t stream_wait_event(uint8_t tid, uint8_t frame_index, uint8_t sid, uint8_t eid)
{
    return hipStreamWaitEvent(d_Streams[tid][sid], d_Events[tid][frame_index][eid],  0);
}


hipError_t stream_signal_event(uint8_t tid, uint8_t frame_index, uint8_t sid, uint8_t eid)
{
    return hipEventRecord(d_Events[tid][frame_index][eid], d_Streams[tid][sid]);
}


hipError_t synchronize_event(uint8_t tid, uint8_t frame_index, uint8_t eid)
{
    return hipEventSynchronize(d_Events[tid][frame_index][eid]);
}

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <CUDA/include/util.h>
#include <CUDA/include/frame_resources.h>

#include <Util/include/debug.h>

#include <stdio.h>


extern "C" void cuda_reset_device()
{
	hipDeviceReset();
}


extern "C" void cuda_device_synchronize()
{
  	hipDeviceSynchronize();
}


extern "C" void cuda_runtime_version(int &major, int &minor)
{
    int runtime_version;
	hipError_t err = hipRuntimeGetVersion(&runtime_version);
	if (err != hipSuccess)
	{
		debug::error("Unable to query CUDA runtime version! Is an Nvidia runtime installed?");
		return;
	}

	major = runtime_version / 1000;
	minor = (runtime_version % 100) / 10; // same as in deviceQuery sample
	if (major < 5 || (major == 5 && minor < 5))
	{
		debug::error("Runtime does not support CUDA 5.5 API! Update your Nvidia runtime!");
		return;
	}
}


extern "C" void cuda_driver_version(int &major, int &minor)
{
	int driver_version;
	hipError_t err = hipDriverGetVersion(&driver_version);
	if (err != hipSuccess)
	{
		debug::error("Unable to query CUDA driver version! Is an Nvidia driver installed?");
		return;
	}

	major = driver_version / 1000;
	minor = (driver_version % 100) / 10; // same as in deviceQuery sample
	if (major < 5 || (major == 5 && minor < 5))
	{
		debug::error("Driver does not support CUDA 5.5 API! Update your Nvidia driver!");
		return;
	}
}


extern "C" uint32_t cuda_device_multiprocessors(uint32_t index)
{
    hipDeviceProp_t props;

	if (hipGetDeviceProperties(&props, index) == hipSuccess)
		return props.multiProcessorCount;

    return 0;
}


extern "C" uint32_t cuda_device_threads(uint32_t index)
{
    hipDeviceProp_t props;

	if(hipGetDeviceProperties(&props, index) == hipSuccess)
    {

        uint32_t threadsPerSM = 0;

        switch (props.major)
        {
            case 3:
            {
                threadsPerSM = 192;
                break;
            }
            case 5:
            {
                threadsPerSM  = 128;
                break;
            }
            case 6:
            {
                if(props.minor == 0)
                    threadsPerSM = 64;
                else
                    threadsPerSM = 128;

                break;
            }
            case 7:
            {
                threadsPerSM = 64;
                break;
            }
            default:
            {
                debug::error(FUNCTION, "GPU #", index,
                    " unsupported compute capability: ", props.major, ".", props.minor);

                return 0;
            }
        }

        return threadsPerSM * props.multiProcessorCount;
    }


    return 0;
}


extern "C" uint32_t cuda_num_devices()
{
    int32_t GPU_N;
    hipError_t err = hipGetDeviceCount(&GPU_N);
    if (err != hipSuccess)
    {
        debug::log(0, "Unable to query number of CUDA devices! Is an nVidia driver installed?");
        return 0;
    }

    return static_cast<uint32_t>(GPU_N);
}


extern "C" std::string cuda_devicename(uint32_t index)
{
	hipDeviceProp_t props;

	if (hipGetDeviceProperties(&props, index) == hipSuccess)
		return std::string(props.name);

	return std::string();
}


extern "C" void cuda_init(uint32_t thr_id)
{
    hipSetDevice(thr_id);
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    hipProfilerStart();
}


extern "C" void cuda_free(uint32_t thr_id)
{
    debug::log(0, "Device ", thr_id, " shutting down...");

    hipSetDevice(thr_id);
    hipDeviceSynchronize();
    hipDeviceReset();
}


extern "C" void cuda_profiler_start()
{
	hipProfilerStart();
}


extern "C" void cuda_profiler_stop()
{
	hipProfilerStop();
}

#include "hip/hip_runtime.h"
/*******************************************************************************************

 Nexus Earth 2018

 (credits: cbuchner1 for sieving)

 [Scale Indefinitely] BlackJack. http://www.opensource.org/licenses/mit-license.php

*******************************************************************************************/

#include <CUDA/include/macro.h>
#include <CUDA/include/combo_sieve.h>
#include <CUDA/include/frame_resources.h>
#include <CUDA/include/sieve_resources.h>
#include <CUDA/include/prime_helper.cuh>
#include <CUDA/include/constants.h>
#include <Util/include/prime_config.h>


template<uint8_t o>
__global__ void combosieve_kernelD_512(uint32_t *g_sieve_hierarchy,
                                       uint32_t *g_bit_array_sieve,
                                       uint32_t *prime_remainders,
                                       uint16_t *blockoffset_mod_p,
                                       uint32_t base_index,
                                       uint16_t nPrimorialEndPrime,
                                       uint16_t nPrimeLimitA)
{
    extern __shared__ uint32_t shared_array_sieve[];
    uint32_t index;
    uint32_t mask;
    uint32_t id;
    uint16_t i, j;
    uint16_t pr, pre2;

    #pragma unroll 16
    for (uint8_t i= 0; i <  16; ++i)
        shared_array_sieve[threadIdx.x + (i << 9)] = 0;

    __syncthreads();

    base_index = base_index << 12;

    //precompute
    uint32_t b_idx = blockIdx.x << 12;

    for (i = nPrimorialEndPrime + threadIdx.x; i < nPrimeLimitA; i += blockDim.x)
    {
        pr = c_primes[i];
        pre2 = blockoffset_mod_p[b_idx + i];


        index = prime_remainders[((base_index + i) << 3) + o] + pre2;  // << 3 because we have space for 8 offsets
        if(index >= pr)
            index = index - pr;

        for(; index < 262144; index += pr)
        {
            mask = 1 << (index & 31);
            id = index >> 5;

            if((g_sieve_hierarchy[id] & mask) == 0)
                atomicOr(&shared_array_sieve[id], mask);
        }

    }

    __syncthreads();
    g_bit_array_sieve += (blockIdx.x << 13);

    #pragma unroll 16
    for (uint8_t i = 0; i < 16; ++i) // fixed value
    {
        j = threadIdx.x + (i << 9);
        //atomicOr(&g_bit_array_sieve[j], shared_array_sieve[j]);
        g_bit_array_sieve[j] = shared_array_sieve[j];
    }
}



template<uint8_t o>
__global__ void combosieve_kernelA_512(uint32_t *g_sieve_hierarchy,
                                       uint32_t *g_bit_array_sieve,
                                       uint32_t *prime_remainders,
                                       uint16_t *blockoffset_mod_p,
                                       uint32_t base_index,
                                       uint16_t nPrimorialEndPrime,
                                       uint16_t nPrimeLimitA)
{
    extern __shared__ uint32_t shared_array_sieve[];
    uint32_t nAdd;
    uint32_t index;
    uint32_t mask;
    uint32_t id;
    uint16_t i, j;
    uint16_t pr, pre2;

    #pragma unroll 16
    for (uint8_t i= 0; i <  16; ++i)
        shared_array_sieve[threadIdx.x + (i << 9)] = 0;

    __syncthreads();

    base_index = base_index << 9;

    for (i = nPrimorialEndPrime; i < nPrimeLimitA; ++i)
    {
        pr = c_primes[i];
        pre2 = blockoffset_mod_p[(blockIdx.x << 12) + i];

        // precompute
        nAdd = pr << 9;

        index = prime_remainders[((base_index + i) << 3) + o] + pre2;  // << 3 because we have space for 8 offsets
        if(index >= pr)
            index = index - pr;

        index = threadIdx.x * pr + index;

        for(; index < 262144; index += nAdd)
        {
            mask = 1 << (index & 31);
            id = index >> 5;

            if((g_sieve_hierarchy[id] & mask) == 0)
                atomicOr(&shared_array_sieve[id], mask);
        }

    }

    __syncthreads();
    g_bit_array_sieve += (blockIdx.x << 13);

    #pragma unroll 16
    for (uint8_t i = 0; i < 16; ++i) // fixed value
    {
        j = threadIdx.x + (i << 9);
        //atomicOr(&g_bit_array_sieve[j], shared_array_sieve[j]);
        g_bit_array_sieve[j] = shared_array_sieve[j];
    }
}


template<uint8_t o>
__global__ void combosieve_kernelB(uint64_t *origins,
                                   uint32_t origin_index,
                                   uint32_t *g_sieve_hierarchy,
                                   uint32_t *bit_array_sieve,
                                   uint32_t bit_array_size,
                                   uint4 *primes,
                                   uint32_t *base_remainders,
                                   uint16_t nPrimorialEndPrime,
                                   uint32_t nPrimeLimit)
{
    uint32_t i = nPrimorialEndPrime + blockDim.x * blockIdx.x + threadIdx.x;

    if(i < nPrimeLimit)
    {
        uint4 tmp = primes[i];
        uint64_t recip = make_uint64_t(tmp.z, tmp.w);
        uint32_t index;
        uint32_t mask;

        tmp.z = mod_p_small(origins[origin_index] + base_remainders[i] + c_offsets[c_iB[o]], tmp.x, recip);
        tmp.w = mod_p_small((uint64_t)(tmp.x - tmp.z)*tmp.y, tmp.x, recip);

        for(; tmp.w < bit_array_size; tmp.w += tmp.x)
        {
            index = tmp.w >> 5;
            mask = c_mark_mask[tmp.w & 31];

            if((g_sieve_hierarchy[index] & mask) == 0)
                atomicOr(&bit_array_sieve[index], mask);
        }
    }
}


__global__ void compact_combo(uint64_t *d_origins,
                              uint64_t *d_nonce_offsets,
                              uint32_t *d_nonce_meta,
                              uint32_t *d_nonce_count,
                              uint32_t *d_bit_array_sieve_A,
                              uint32_t *d_bit_array_sieve_B,
                              uint32_t nBitArray_Size,
                              uint32_t origin_index,
                              uint32_t nMaxCandidates,
                              uint8_t nThreshold,
                              uint8_t nOffsetsB,
                              uint8_t nOffsets)
{
    /* If the quit flag was set, early return to avoid wasting time. */
    if(c_quit)
    {
        *d_nonce_count = 0;
        return;
    }


    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    //uint32_t sharedSizeBits = 32 * 1024 * 8;
    //uint32_t allocSize = ((nBitArray_Size + sharedSizeBits-1) / sharedSizeBits) * sharedSizeBits;
    //uint32_t nSieveWords = (allocSize + 31) >> 5;

    if(idx < nBitArray_Size)
    {
        uint64_t nonce_offset = c_primorial * (uint64_t)idx + d_origins[origin_index];
        uint32_t combo = 0;

        uint32_t index = idx >> 5;
        uint32_t mask = c_mark_mask[idx & 31];
        uint32_t nBitArray_Words = nBitArray_Size >> 5;

        /* Check the single sieve to see if offsets are valid. */
        if((d_bit_array_sieve_A[index] & mask) == 0)
        {
            combo = c_bitmaskA;

            /* Take the bit from each sieve and compact them into a single word. */
            for(uint8_t o = 0; o < nOffsetsB; ++o)
            {
                /* Use logical not operator to reduce result into inverted 0 or 1 bit. */
                uint32_t bit = !(d_bit_array_sieve_B[index] & mask);

                combo |= bit << c_iB[o];

                d_bit_array_sieve_B += nBitArray_Words;
            }

            /* Get the count of the remaining zero bits and compare to threshold. */
            uint32_t nRemaining = __popc(combo);

            /* Count the remaining bits for this combo. */
            if(nRemaining >= nThreshold)
            {
                //printf("%d: compact_sieve: combo=%08X, count=%d\n", idx, combo, nRemaining);

                /* Compute the end, tail, and next indices. */
                //uint32_t e = 32 - __clz(combo);
                uint32_t t = 0;
                uint32_t n = __ffs(combo);

                /* Iterate through sieved bits and determine if there is a rule-breaking prime gap. */
                uint32_t nCount = 1;
                for(; nCount < nRemaining; ++nCount)
                {
                    t = n;
                    n = n + __ffs(combo >> (n+1) );

                    if( (c_offsets[n] - c_offsets[t]) > 12)
                        break;
                }

                /* Determine if combo count will break threshold. */
                if(nCount >= nThreshold)
                {
                    uint32_t i = atomicAdd(d_nonce_count, 1);

                    if(i < nMaxCandidates)
                    {
                        /* Assign the global nonce offset and meta data. */
                        d_nonce_offsets[i] = nonce_offset;
                        d_nonce_meta[i] = ~combo;
                    }
                }
            }
        }
    }
}


#define COMBO_A_LAUNCH(X) combosieve_kernelD_512<X><<<grid, block, sharedSizeBits/8, d_Streams[thr_id][str_id]>>>(\
frameResources[thr_id].d_bit_array_sieve[frame_index], \
&frameResources[thr_id].d_bit_array_sieve[frame_index][X * nBitArray_Size >> 5], \
&d_prime_remainders[thr_id][4096 * nOrigins << 3], \
d_blockoffset_mod_p[thr_id], \
origin_index, \
nPrimorialEndPrime, \
nPrimeLimitA)


void comboA_launch(uint8_t thr_id,
                   uint8_t str_id,
                   uint32_t origin_index,
                   uint8_t frame_index,
                   uint16_t nPrimorialEndPrime,
                   uint16_t nPrimeLimitA,
                   uint32_t nBitArray_Size,
                   uint32_t nOrigins)
{
    uint32_t sharedSizeBits = 32 * 1024 * 8;
    uint32_t nBlocks = (nBitArray_Size + sharedSizeBits-1) / sharedSizeBits;

    //uint32_t allocSize = ((nBitArray_Size + sharedSizeBits-1) / sharedSizeBits) * sharedSizeBits;
    //uint32_t nSieveWords = (allocSize + 31) >> 5;

    dim3 block(512);
    dim3 grid(nBlocks);

    /* fall-through switch logic, zero-based indexing */
    switch(nOffsetsB)
    {
        case 8:  COMBO_A_LAUNCH(8);
        case 7:  COMBO_A_LAUNCH(7);
        case 6:  COMBO_A_LAUNCH(6);
        case 5:  COMBO_A_LAUNCH(5);
        case 4:  COMBO_A_LAUNCH(4);
        case 3:  COMBO_A_LAUNCH(3);
        case 2:  COMBO_A_LAUNCH(2);
        case 1:  COMBO_A_LAUNCH(1);
        break;
        default: debug::error("Unsupported Combo A Launch.");
        break;
    }

    debug::log(4, FUNCTION, (uint32_t)thr_id);
}

#define COMBO_B_LAUNCH(X)   combosieve_kernelB<X><<<grid, block, 0, d_Streams[thr_id][str_id]>>>( \
d_origins[thr_id], \
origin_index, \
frameResources[thr_id].d_bit_array_sieve[frame_index], \
&frameResources[thr_id].d_bit_array_sieve[frame_index][X * nBitArray_Size >> 5], \
nBitArray_Size, \
d_primesInverseInvk[thr_id], \
d_base_remainders[thr_id], \
nPrimorialEndPrime, \
nPrimeLimit)

void comboB_launch(uint8_t thr_id,
                    uint8_t str_id,
                    uint32_t origin_index,
                    uint8_t frame_index,
                    uint16_t nPrimorialEndPrime,
                    uint32_t nPrimeLimit,
                    uint32_t nBitArray_Size)
{
    uint32_t nThreads = nPrimeLimit - nPrimorialEndPrime;
    uint32_t nThreadsPerBlock = 512;
    uint32_t nBlocks = (nThreads + nThreadsPerBlock - 1) / nThreadsPerBlock;

    dim3 block(nThreadsPerBlock);
    dim3 grid(nBlocks);

    /* fall-through switch logic, zero-based indexing */
    switch(nOffsetsB)
    {
        case 8:  COMBO_B_LAUNCH(8);
        case 7:  COMBO_B_LAUNCH(7);
        case 6:  COMBO_B_LAUNCH(6);
        case 5:  COMBO_B_LAUNCH(5);
        case 4:  COMBO_B_LAUNCH(4);
        case 3:  COMBO_B_LAUNCH(3);
        case 2:  COMBO_B_LAUNCH(2);
        case 1:  COMBO_B_LAUNCH(1);
        break;
        default: debug::error("Unsupported Combo B Launch.");
        break;
    }

    debug::log(4, FUNCTION, (uint32_t)thr_id);
}

#define COMBO_COMPACT_LAUNCH(X) compact_combo<<<grid, block, 0, d_Streams[thr_id][str_id]>>>( \
    d_origins[thr_id], \
    frameResources[thr_id].d_pre_nonce_offsets[curr_test], \
    frameResources[thr_id].d_pre_nonce_meta[curr_test], \
    frameResources[thr_id].d_pre_nonce_count[curr_test], \
    frameResources[thr_id].d_bit_array_sieve[curr_sieve], \
    &frameResources[thr_id].d_bit_array_sieve[curr_sieve][nBitArray_Size >> 5], \
    nBitArray_Size, \
    origin_index, \
    nMaxCandidates, \
    threshold, \
    X, \
    vOffsets.size())


void kernel_ccompact_launch(uint8_t thr_id,
                            uint8_t str_id,
                            uint32_t origin_index,
                            uint32_t nMaxCandidates,
                            uint8_t curr_sieve,
                            uint8_t curr_test,
                            uint8_t next_test,
                            uint32_t nBitArray_Size,
                            uint8_t threshold)
{
    dim3 block(64);
    dim3 grid((nBitArray_Size + block.x - 1) / block.x);

    switch(nOffsetsB)
    {
        case 1:  COMBO_COMPACT_LAUNCH(1);  break;
        case 2:  COMBO_COMPACT_LAUNCH(2);  break;
        case 3:  COMBO_COMPACT_LAUNCH(3);  break;
        case 4:  COMBO_COMPACT_LAUNCH(4);  break;
        case 5:  COMBO_COMPACT_LAUNCH(5);  break;
        case 6:  COMBO_COMPACT_LAUNCH(6);  break;
        case 7:  COMBO_COMPACT_LAUNCH(7);  break;
        case 8:  COMBO_COMPACT_LAUNCH(8);  break;
        default: debug::error("Unsupported Combo Compact Launch."); break;
    }

    /* Copy the nonce count for this compaction. */
    CHECK(hipMemcpyAsync(
            frameResources[thr_id].h_nonce_count[curr_test],
            frameResources[thr_id].d_pre_nonce_count[curr_test],
            sizeof(uint32_t), hipMemcpyDeviceToHost, d_Streams[thr_id][str_id]));

    /*Prepare empty initial count for next compaction buffer. */
    *frameResources[thr_id].h_nonce_count[next_test] = 0;

    CHECK(hipMemcpyAsync(
            frameResources[thr_id].d_pre_nonce_count[next_test],
            frameResources[thr_id].h_nonce_count[next_test],
            sizeof(uint32_t), hipMemcpyHostToDevice, d_Streams[thr_id][str_id]));

    CHECK(hipMemcpyAsync(
            frameResources[thr_id].d_nonce_count[next_test],
            frameResources[thr_id].h_nonce_count[next_test],
            sizeof(uint32_t), hipMemcpyHostToDevice, d_Streams[thr_id][str_id]));

    debug::log(4, FUNCTION, (uint32_t)thr_id);
}

#include "hip/hip_runtime.h"
/*******************************************************************************************

 Nexus Earth 2018

 (credits: cbuchner1 for sieving)

 [Scale Indefinitely] BlackJack. http://www.opensource.org/licenses/mit-license.php

*******************************************************************************************/

#include <CUDA/include/macro.h>
#include <CUDA/include/sieve.h>
#include <CUDA/include/util.h>
#include <CUDA/include/frame_resources.h>
#include <CUDA/include/sieve_resources.h>
#include <CUDA/include/streams_events.h>

#include <CUDA/include/combo_sieve.h>

#include <CUDA/include/constants.h>
#include <CUDA/include/unroller.cuh>

#include <Util/include/debug.h>

#include <hip/hip_runtime.h>
#include <stdio.h>



struct FrameResource frameResources[GPU_MAX];
uint4 *d_primesInverseInvk[GPU_MAX];
uint64_t *d_origins[GPU_MAX];
uint32_t *d_primes[GPU_MAX];
uint32_t *d_prime_remainders[GPU_MAX];
uint32_t *d_base_remainders[GPU_MAX];
uint16_t *d_blockoffset_mod_p[GPU_MAX];
uint32_t nOffsetsA;
uint32_t nOffsetsB;


extern "C" void cuda_set_offset_patterns(uint8_t thr_id,
                                         const std::vector<uint32_t> &offsets,
                                         const std::vector<uint32_t> &indicesA,
                                         const std::vector<uint32_t> &indicesB,
                                         const std::vector<uint32_t> &indicesT)
{
    nOffsetsA = indicesA.size();
    nOffsetsB = indicesB.size();
    uint32_t nOffsetsT = indicesT.size();
    uint32_t nOffsets = offsets.size();
    uint32_t bitMaskA = 0;
    uint32_t bitMaskT = 0;

    if(nOffsets > OFFSETS_MAX)
    {
        debug::error(FUNCTION, "Cannot have more than 32 total offsets.");
        return;
    }

    if(nOffsetsA > 8 || nOffsetsB > 8 || nOffsetsT > 8)
    {
        debug::error(FUNCTION, "Cannot have more than 8 offsets");
        return;
    }

    /* Find the start and end indices for all GPU offsets. */
    uint32_t ibeg = 32;
    uint32_t iend = 0;

    for(uint8_t i = 0; i < nOffsetsA; ++i)
    {
        bitMaskA |= (1 << indicesA[i]);
        ibeg = std::min(ibeg, indicesA[i]);
        iend = std::max(iend, indicesA[i]);
    }
    for(uint8_t i = 0; i < nOffsetsB; ++i)
    {
        ibeg = std::min(ibeg, indicesB[i]);
        iend = std::max(iend, indicesB[i]);
    }

    for(uint8_t i = 0; i < nOffsetsT; ++i)
    {
        bitMaskT |= (1 << indicesT[i]);
    }

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_bitmaskT), &bitMaskT,
         sizeof(uint32_t), 0, hipMemcpyHostToDevice));

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_bitmaskA), &bitMaskA,
         sizeof(uint32_t), 0, hipMemcpyHostToDevice));

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_offsets), offsets.data(),
        nOffsets*sizeof(uint32_t), 0, hipMemcpyHostToDevice));

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_iA), indicesA.data(),
        nOffsetsA*sizeof(uint32_t), 0, hipMemcpyHostToDevice));

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_iB), indicesB.data(),
        nOffsetsB*sizeof(uint32_t), 0, hipMemcpyHostToDevice));

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_iT), indicesT.data(),
        nOffsetsT*sizeof(uint32_t), 0, hipMemcpyHostToDevice));

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_iBeg), &ibeg, sizeof(uint32_t), 0, hipMemcpyHostToDevice));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_iEnd), &iend, sizeof(uint32_t), 0, hipMemcpyHostToDevice));
}


extern "C" void cuda_set_zTempVar(uint8_t thr_id, const uint64_t *limbs)
{
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_zTempVar), limbs, 17*sizeof(uint64_t), 0, hipMemcpyHostToDevice));
}


extern "C" void cuda_init_primes(uint8_t thr_id,
                                 uint64_t *origins,
                                 uint32_t *primes,
                                 uint32_t *primesInverseInvk,
                                 uint32_t nPrimeLimit,
                                 uint32_t nBitArray_Size,
                                 uint32_t sharedSizeKB,
                                 uint32_t nPrimorialEndPrime,
                                 uint32_t nPrimeLimitA,
                                 uint32_t nOrigins,
                                 uint32_t nMaxCandidates)
{
    uint32_t primeinverseinvk_size = sizeof(uint32_t) * 4 * nPrimeLimit;
    uint64_t nonce64_size = nMaxCandidates * sizeof(uint64_t);
    uint64_t nonce32_size = nMaxCandidates * sizeof(uint32_t);
    uint32_t sharedSizeBits = sharedSizeKB * 1024 * 8;
    uint32_t allocSize = ((nBitArray_Size * 16  + sharedSizeBits - 1) / sharedSizeBits) * sharedSizeBits;
    uint32_t bitarray_size = (allocSize+31)/32 * sizeof(uint32_t);
    uint32_t remainder_size = 2 * 8 * 4096 * nOrigins * sizeof(uint32_t);

    /* Allocate memory for the primes, inverses, and reciprocals that are used
       as the basis for prime sieve computation */
    CHECK(hipMalloc(&d_primesInverseInvk[thr_id],  primeinverseinvk_size));
    CHECK(hipMemcpy(d_primesInverseInvk[thr_id], primesInverseInvk, primeinverseinvk_size, hipMemcpyHostToDevice));

    /* Allocate base remainders that will be pre-computed once per block */
    CHECK(hipMalloc(&d_base_remainders[thr_id],  nPrimeLimit * sizeof(uint32_t)));

    /* Create list of primes only */
    //CHECK(hipMalloc(&d_primes[thr_id], nPrimeLimit * sizeof(uint32_t)));
    //CHECK(hipMemcpy(d_primes[thr_id], primes, nPrimeLimit * sizeof(uint32_t), hipMemcpyHostToDevice));


    /* Allocate memory for prime origins. */
    CHECK(hipMalloc(&d_origins[thr_id], nOrigins * sizeof(uint64_t)));


    CHECK(hipMalloc(&d_prime_remainders[thr_id], remainder_size));


    /* Allocate multiple frame resources so we can keep multiple frames in flight
       to further improve CPU/GPU utilization */

    for(uint8_t i = 0; i < FRAME_COUNT; ++i)
    {
        /* test */
        CHECK(    hipMalloc(&frameResources[thr_id].d_result_offsets[i], nonce64_size));
        CHECK(hipHostMalloc(&frameResources[thr_id].h_result_offsets[i], nonce64_size));
        CHECK(    hipMalloc(&frameResources[thr_id].d_result_meta[i],    nonce32_size));
        CHECK(hipHostMalloc(&frameResources[thr_id].h_result_meta[i],    nonce32_size));
        CHECK(    hipMalloc(&frameResources[thr_id].d_result_count[i],   sizeof(uint32_t)));
        CHECK(hipHostMalloc(&frameResources[thr_id].h_result_count[i],   sizeof(uint32_t)));

        CHECK(    hipMalloc(&frameResources[thr_id].d_window_data[i], nonce32_size * WORD_MAX * WINDOW_SIZE));

        /* test stats */
        CHECK(    hipMalloc(&frameResources[thr_id].d_primes_checked[i], OFFSETS_MAX * sizeof(uint32_t)));
        CHECK(hipHostMalloc(&frameResources[thr_id].h_primes_checked[i], OFFSETS_MAX * sizeof(uint32_t)));
        CHECK(    hipMalloc(&frameResources[thr_id].d_primes_found[i],   OFFSETS_MAX * sizeof(uint32_t)));
        CHECK(hipHostMalloc(&frameResources[thr_id].h_primes_found[i],   OFFSETS_MAX * sizeof(uint32_t)));

        /* compaction */
        CHECK(    hipMalloc(&frameResources[thr_id].d_nonce_offsets[i], nonce64_size));
        CHECK(    hipMalloc(&frameResources[thr_id].d_nonce_meta[i],    nonce32_size));
        CHECK(    hipMalloc(&frameResources[thr_id].d_nonce_count[i],   sizeof(uint32_t)));

        CHECK(    hipMalloc(&frameResources[thr_id].d_pre_nonce_offsets[i], nonce64_size));
        CHECK(    hipMalloc(&frameResources[thr_id].d_pre_nonce_meta[i],    nonce32_size));
        CHECK(    hipMalloc(&frameResources[thr_id].d_pre_nonce_count[i],   sizeof(uint32_t)));

        CHECK(hipHostMalloc(&frameResources[thr_id].h_nonce_count[i],   sizeof(uint32_t)));

        /* sieving */

        //CHECK(    hipMalloc(&frameResources[thr_id].d_bit_array_sieve[i], bitarray_size));

        /* combo sieve */
        CHECK(    hipMalloc(&frameResources[thr_id].d_bit_array_sieve[i], bitarray_size));


        /* bucket sieve (experimental) */
        // CHECK(hipMalloc(&frameResources[thr_id].d_bucket_o[i], sizeof(uint32_t) * nPrimeLimit << 4));
        // CHECK(hipMalloc(&frameResources[thr_id].d_bucket_away[i], sizeof(uint16_t) * nPrimeLimit << 4));
    }

    /* Have capacity for small primes up to 4096 */
    uint16_t p[4096];
    for(uint32_t i = 0; i < nPrimeLimitA; ++i)
        p[i] = primes[i];

    /* Copy small primes to GPU. */
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_primes), p, nPrimeLimitA * sizeof(uint16_t), 0, hipMemcpyHostToDevice));

    /* Allocate and compute block offsets for a list of small prime mod offsets
       at each block offset in the gpu small sieve kernel */
    uint32_t nBlocks = (nBitArray_Size + sharedSizeBits-1) / sharedSizeBits;
    uint32_t blockoffset_size = nBlocks * 4096 * sizeof(uint16_t);

    CHECK(hipMalloc(&d_blockoffset_mod_p[thr_id], blockoffset_size));

    uint16_t *offsets = (uint16_t *)malloc(blockoffset_size);

    for (uint32_t block = 0; block < nBlocks; ++block)
    {
        uint32_t blockOffset = sharedSizeBits * block;

        for (uint32_t i = 0; i < nPrimeLimitA; ++i)
            offsets[block*4096 + i] = primes[i] - (blockOffset % primes[i]);
    }
    CHECK(hipMemcpy(d_blockoffset_mod_p[thr_id], offsets, blockoffset_size, hipMemcpyHostToDevice));
    free(offsets);

    /* Create the CUDA streams and events used for sieve, compacting, and testing */
    streams_events_init(thr_id);
}

extern "C" void cuda_free_primes(uint8_t thr_id)
{
    CHECK(hipFree(d_primesInverseInvk[thr_id]));
    CHECK(hipFree(d_base_remainders[thr_id]));
    //CHECK(hipFree(d_primes[thr_id]));
    CHECK(hipFree(d_origins[thr_id]));
    CHECK(hipFree(d_prime_remainders[thr_id]));

    for(uint8_t i = 0; i < FRAME_COUNT; ++i)
    {
        CHECK(    hipFree(frameResources[thr_id].d_result_offsets[i]));
        CHECK(hipHostFree(frameResources[thr_id].h_result_offsets[i]));

        CHECK(    hipFree(frameResources[thr_id].d_result_meta[i]));
        CHECK(hipHostFree(frameResources[thr_id].h_result_meta[i]));

        CHECK(    hipFree(frameResources[thr_id].d_result_count[i]));
        CHECK(hipHostFree(frameResources[thr_id].h_result_count[i]));

        CHECK(    hipFree(frameResources[thr_id].d_primes_checked[i]));
        CHECK(hipHostFree(frameResources[thr_id].h_primes_checked[i]));

        CHECK(    hipFree(frameResources[thr_id].d_primes_found[i]));
        CHECK(hipHostFree(frameResources[thr_id].h_primes_found[i]));

        CHECK(    hipFree(frameResources[thr_id].d_nonce_offsets[i]));
        CHECK(    hipFree(frameResources[thr_id].d_nonce_meta[i]));
        CHECK(    hipFree(frameResources[thr_id].d_nonce_count[i]));

        CHECK(    hipFree(frameResources[thr_id].d_pre_nonce_offsets[i]));
        CHECK(    hipFree(frameResources[thr_id].d_pre_nonce_meta[i]));
        CHECK(    hipFree(frameResources[thr_id].d_pre_nonce_count[i]));

        CHECK(hipHostFree(frameResources[thr_id].h_nonce_count[i]));


        CHECK(    hipFree(frameResources[thr_id].d_bit_array_sieve[i]));

        //CHECK(hipFree(frameResources[thr_id].d_bucket_o[i]));
        //CHECK(hipFree(frameResources[thr_id].d_bucket_away[i]));
    }

    CHECK(hipFree(d_blockoffset_mod_p[thr_id]));

    streams_events_free(thr_id);
}


__global__ void base_remainders_kernel(uint4 *primes, uint32_t *base_remainders, uint32_t nPrimeLimit)
{
    uint32_t i = (blockDim.x * blockIdx.x + threadIdx.x);


    if (i < nPrimeLimit)
    {
        uint4 tmp = primes[i];
        base_remainders[i] = mpi_mod_int(c_zTempVar, tmp.x, make_uint64_t(tmp.z, tmp.w));
    }

}

extern "C" void cuda_base_remainders(uint8_t thr_id, uint32_t nPrimeLimit)
{
    int nThreadsPerBlock = 32;

    for(uint8_t i = 0; i < 4; ++i)
    {

        dim3 block(nThreadsPerBlock);

        int i_beg = i * nPrimeLimit / 4;
        int i_end = (i+1) * nPrimeLimit / 4;

        if(i == 3)
            i_end = nPrimeLimit;

        int nThreads = i_end - i_beg;
        int nBlocks = (nThreads + nThreadsPerBlock - 1) / nThreadsPerBlock;
        dim3 grid(nBlocks);

        CHECK(stream_wait_event(thr_id, 0, i, i));

        base_remainders_kernel<<<grid, block, 0, d_Streams[thr_id][i]>>>(&d_primesInverseInvk[thr_id][i_beg],
                                                &d_base_remainders[thr_id][i_beg],
                                                nThreads);

        CHECK(stream_signal_event(thr_id, 0, i, i));

    }
}


__global__ void primesieve_kernelA0(uint64_t *origins,
                                    uint4 *primes,
                                    uint32_t *prime_remainders,
                                    uint32_t *base_remainders,
                                    uint8_t nOffsets,
                                    uint16_t nOrigins,
                                    uint32_t nThreads)
{
    uint32_t g_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(g_idx < nThreads)
    {
        uint32_t p_idx = g_idx / nOrigins;
        uint64_t o_idx = g_idx % nOrigins;

        uint32_t j = ((o_idx << 12) + p_idx) << 3;


        o_idx = origins[o_idx] + base_remainders[p_idx];

        uint4 tmp = primes[p_idx];
        uint64_t recip = make_uint64_t(tmp.z, tmp.w);

        tmp.z = mod_p_small(o_idx, tmp.x, recip);

        uint32_t pr;

        for(uint8_t o = 0; o < nOffsets; ++o)
        {
            pr = tmp.z + c_offsets[c_iA[o]];
            if(pr >= tmp.x)
                pr -= tmp.x;

            prime_remainders[j + o] = mod_p_small((uint64_t)(tmp.x - pr)*tmp.y, tmp.x, recip);
        }
    }
}


__global__ void primesieve_kernelB0(uint64_t *origins,
                                    uint4 *primes,
                                    uint32_t *prime_remainders,
                                    uint32_t *base_remainders,
                                    uint8_t nOffsets,
                                    uint16_t nOrigins,
                                    uint32_t nThreads)
{
    uint32_t g_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(g_idx < nThreads)
    {
        uint32_t p_idx = g_idx / nOrigins;
        uint64_t o_idx = g_idx % nOrigins;

        uint32_t j = ((o_idx << 12) + p_idx) << 3;


        o_idx = origins[o_idx] + base_remainders[p_idx];

        uint4 tmp = primes[p_idx];
        uint64_t recip = make_uint64_t(tmp.z, tmp.w);

        tmp.z = mod_p_small(o_idx, tmp.x, recip);

        uint32_t pr;

        for(uint8_t o = 0; o < nOffsets; ++o)
        {
            pr = tmp.z + c_offsets[c_iB[o]];
            if(pr >= tmp.x)
                pr -= tmp.x;

            prime_remainders[j + o] = mod_p_small((uint64_t)(tmp.x - pr)*tmp.y, tmp.x, recip);
        }
    }
}


template<uint8_t offsetsA>
__global__ void primesieve_kernelA_512(uint32_t *g_bit_array_sieve,
                                       uint32_t *prime_remainders,
                                       uint16_t *blockoffset_mod_p,
                                       uint32_t base_index,
                                       uint16_t nPrimorialEndPrime,
                                       uint16_t nPrimeLimitA)
{
    extern __shared__ uint32_t shared_array_sieve[];
    uint32_t pIdx;
    uint32_t nAdd;
    uint32_t pre1[offsetsA];
    uint32_t index;
    uint16_t i, j;
    uint16_t pr, pre2;

    #pragma unroll 16
    for (int i= 0; i <  16; ++i)
        shared_array_sieve[threadIdx.x + (i << 9)] = 0;

    __syncthreads();

    base_index = base_index << 12;

    for (i = nPrimorialEndPrime; i < nPrimeLimitA; ++i)
    {
        pr = c_primes[i];
        pre2 = blockoffset_mod_p[(blockIdx.x << 12) + i];

        // precompute
        pIdx = threadIdx.x * pr;
        nAdd = (base_index + i) << 3;

        auto pre = [&pre1, &prime_remainders, &nAdd](uint32_t o)
        {
            pre1[o] = prime_remainders[nAdd + o]; // << 3 because we have space for 8 offsets
        };

        Unroller<0, offsetsA>::step(pre);

        nAdd = pr << 9;
        auto loop = [&pIdx, &nAdd, &pre1, &pre2, &pr, &index](uint32_t o)
        {
            index = pre1[o] + pre2;
            if(index >= pr)
                index = index - pr;

            index = index + pIdx;

            for(; index < 262144; index += nAdd)
            {
                atomicOr(&shared_array_sieve[index >> 5], 1 << (index & 31));
            }
        };

        Unroller<0, offsetsA>::step(loop);
    }

    __syncthreads();
    g_bit_array_sieve += (blockIdx.x << 13);

    #pragma unroll 16
    for (int i = 0; i < 16; ++i) // fixed value
    {
        j = threadIdx.x + (i << 9);
        //atomicOr(&g_bit_array_sieve[j], shared_array_sieve[j]);
        g_bit_array_sieve[j] = shared_array_sieve[j];
    }
}


template<uint8_t offsetsA>
__global__ void primesieve_kernelD_512(uint32_t *g_bit_array_sieve,
                                       uint32_t *prime_remainders,
                                       uint16_t *blockoffset_mod_p,
                                       uint32_t base_index,
                                       uint16_t nPrimorialEndPrime,
                                       uint16_t nPrimeLimitA)
{
    extern __shared__ uint32_t shared_array_sieve[];
    uint32_t pre1[offsetsA];
    uint32_t index;
    uint16_t i, j;
    uint16_t pr, pre2;

    #pragma unroll 16
    for (int i= 0; i <  16; ++i)
        shared_array_sieve[threadIdx.x + (i << 9)] = 0;

    __syncthreads();

    base_index = base_index << 12;

    //precompute
    uint32_t b_idx = blockIdx.x << 12;

    auto pre = [&pre1, &prime_remainders, &index](uint32_t o)
    {
        pre1[o] = prime_remainders[index + o]; // << 3 because we have space for 8 offsets
    };

    auto loop = [&pre1, &pre2, &pr, &index](uint32_t o)
    {
        index = pre1[o] + pre2;
        if(index >= pr)
            index = index - pr;

        for(; index < 262144; index += pr)
        {
            atomicOr(&shared_array_sieve[index >> 5], 1 << (index & 31));
        }
    };

    for (i = nPrimorialEndPrime + threadIdx.x; i < nPrimeLimitA; i += blockDim.x)
    {
        pr = c_primes[i];
        pre2 = blockoffset_mod_p[b_idx + i];

        // precompute
        index = (base_index + i) << 3;

        Unroller<0, offsetsA>::step(pre);
        Unroller<0, offsetsA>::step(loop);
    }

    __syncthreads();
    g_bit_array_sieve += (blockIdx.x << 13);

    #pragma unroll 16
    for (int i = 0; i < 16; ++i) // fixed value
    {
        j = threadIdx.x + (i << 9);
        //atomicOr(&g_bit_array_sieve[j], shared_array_sieve[j]);
        g_bit_array_sieve[j] = shared_array_sieve[j];
    }
}


__global__ void clearsieve_kernel(uint32_t *sieve, uint32_t n_words)
{
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < n_words)
        sieve[i] = 0;
}


__global__ void primesieve_kernelB(uint64_t *origins,
                                   uint32_t *bit_array_sieve,
                                   uint32_t bit_array_size,
                                   uint4 *primes,
                                   uint32_t *base_remainders,
                                   uint32_t nPrimorialEndPrime,
                                   uint32_t nPrimeLimit,
                                   uint32_t nOffsets,
                                   uint32_t origin_index)
{
    uint32_t i = nPrimorialEndPrime + blockDim.x * blockIdx.x + threadIdx.x;

    if(i < nPrimeLimit)
    {
        uint4 tmp = primes[i];
        uint64_t origin = origins[origin_index] + base_remainders[i];
        uint64_t recip = make_uint64_t(tmp.z, tmp.w);

        uint32_t index;
        uint32_t mask;

        tmp.z = mod_p_small(origin, tmp.x, recip);

        for(uint32_t o = 0; o < nOffsets; ++o)
        {
            index = tmp.z + c_offsets[c_iA[o]];
            if(index >= tmp.x)
                index -= tmp.x;

            tmp.w = mod_p_small((uint64_t)(tmp.x - index)*tmp.y, tmp.x, recip);


            for(; tmp.w < bit_array_size; tmp.w += tmp.x)
            {
                index = tmp.w >> 5;
                mask = c_mark_mask[tmp.w & 31];

                if((bit_array_sieve[index] & mask) == 0)
                    atomicOr(&bit_array_sieve[index], mask);
            }
        }
    }
}


__global__ void primesieve_kernelC(uint64_t *origins,
                                   uint32_t *bit_array_sieve,
                                   uint32_t bit_array_size,
                                   uint4 *primes,
                                   uint32_t *base_remainders,
                                   uint32_t nPrimorialEndPrime,
                                   uint32_t nPrimeLimit,
                                   uint32_t nOffsets,
                                   uint32_t origin_index)
{
    uint32_t i = nPrimorialEndPrime + blockDim.x * blockIdx.x + threadIdx.x;

    if(i < nPrimeLimit)
    {
        uint4 tmp = primes[i];
        uint64_t origin = origins[origin_index] + base_remainders[i];
        uint64_t recip = make_uint64_t(tmp.z, tmp.w);

        uint32_t index;
        uint32_t mask;

        tmp.z = mod_p_small(origin, tmp.x, recip);

        for(uint32_t o = 0; o < nOffsets; ++o)
        {
            index = tmp.z + c_offsets[c_iA[o]];
            if(index >= tmp.x)
                index -= tmp.x;

            tmp.w = mod_p_small((uint64_t)(tmp.x - index)*tmp.y, tmp.x, recip);


            if(tmp.w < bit_array_size)
            {
                index = tmp.w >> 5;
                mask = c_mark_mask[tmp.w & 31];

                if((bit_array_sieve[index] & mask) == 0)
                    atomicOr(&bit_array_sieve[index], mask);
            }
        }
    }
}


void kernelA0_launch(uint8_t thr_id,
                     uint8_t str_id,
                     uint16_t nPrimeLimitA,
                     uint16_t nOrigins)
{
    uint32_t nThreads = nPrimeLimitA * nOrigins;


    dim3 block(512);
    dim3 grid((nThreads + block.x - 1) / block.x);

    primesieve_kernelA0<<<grid, block, 0, d_Streams[thr_id][str_id]>>>(
        d_origins[thr_id],
        d_primesInverseInvk[thr_id],
        d_prime_remainders[thr_id],
        d_base_remainders[thr_id],
        nOffsetsA,
        nOrigins,
        nThreads);
}

void kernelB0_launch(uint8_t thr_id,
                     uint8_t str_id,
                     uint16_t nPrimeLimitA,
                     uint16_t nOrigins)
{
    uint32_t nThreads = nPrimeLimitA * nOrigins;

    dim3 block(512);
    dim3 grid((nThreads + block.x - 1) / block.x);

    primesieve_kernelB0<<<grid, block, 0, d_Streams[thr_id][str_id]>>>(
        d_origins[thr_id],
        d_primesInverseInvk[thr_id],
        &d_prime_remainders[thr_id][4096 * nOrigins << 3],
        d_base_remainders[thr_id],
        nOffsetsB,
        nOrigins,
        nThreads);
}

extern "C" void cuda_set_origins(uint8_t thr_id, uint32_t nPrimeLimitA, uint64_t *origins, uint32_t nOrigins)
{
    CHECK(hipMemcpy(d_origins[thr_id], origins, nOrigins * sizeof(uint64_t), hipMemcpyHostToDevice));


    /* Precompute prime remainders. */

    debug::log(4, FUNCTION, "nPrimeLimitA=", nPrimeLimitA, " nOrigins=", nOrigins);

    for(uint8_t i = 0; i < 4; ++i)
        CHECK(stream_wait_event(thr_id, 0, i, i));

    kernelA0_launch(thr_id, STREAM::SIEVE_A, nPrimeLimitA, nOrigins);
    kernelB0_launch(thr_id, STREAM::SIEVE_B, nPrimeLimitA, nOrigins);

    CHECK(stream_signal_event(thr_id, 0, STREAM::SIEVE_A, EVENT::SIEVE_A));
    CHECK(stream_signal_event(thr_id, 0, STREAM::SIEVE_B, EVENT::SIEVE_B));
}

#define KERNEL_A_LAUNCH(X) primesieve_kernelD_512<X><<<grid, block, sharedSizeBits/8, d_Streams[thr_id][str_id]>>>(\
frameResources[thr_id].d_bit_array_sieve[frame_index], \
d_prime_remainders[thr_id], \
d_blockoffset_mod_p[thr_id], \
origin_index, \
nPrimorialEndPrime, \
nPrimeLimitA)

void kernelA_launch(uint8_t thr_id,
                    uint8_t str_id,
                    uint32_t origin_index,
                    uint8_t frame_index,
                    uint16_t nPrimorialEndPrime,
                    uint16_t nPrimeLimitA,
                    uint32_t nBitArray_Size)
{
    const int sharedSizeBits = 32 * 1024 * 8;
    int nBlocks = (nBitArray_Size + sharedSizeBits-1) / sharedSizeBits;

    dim3 block(512);
    dim3 grid(nBlocks);

    switch(nOffsetsA)
    {
        case 1:  KERNEL_A_LAUNCH(1);  break;
        case 2:  KERNEL_A_LAUNCH(2);  break;
        case 3:  KERNEL_A_LAUNCH(3);  break;
        case 4:  KERNEL_A_LAUNCH(4);  break;
        case 5:  KERNEL_A_LAUNCH(5);  break;
        case 6:  KERNEL_A_LAUNCH(6);  break;
        case 7:  KERNEL_A_LAUNCH(7);  break;
        case 8:  KERNEL_A_LAUNCH(8);  break;
        case 9:  KERNEL_A_LAUNCH(9);  break;
        case 10: KERNEL_A_LAUNCH(10); break;
        case 11: KERNEL_A_LAUNCH(11); break;
        case 12: KERNEL_A_LAUNCH(12); break;
        case 13: KERNEL_A_LAUNCH(13); break;
        case 14: KERNEL_A_LAUNCH(14); break;
        case 15: KERNEL_A_LAUNCH(15); break;
    }
}


#define KERNEL_B_LAUNCH(X)   primesieve_kernelB<<<grid, block, 0, d_Streams[thr_id][str_id]>>>( \
d_origins[thr_id], \
frameResources[thr_id].d_bit_array_sieve[frame_index], \
nBitArray_Size, \
d_primesInverseInvk[thr_id], \
d_base_remainders[thr_id], \
nPrimeLimitA, \
nPrimeLimitB, \
X, \
origin_index )

void kernelB_launch(uint8_t thr_id,
                    uint8_t str_id,
                    uint32_t origin_index,
                    uint8_t frame_index,
                    uint32_t nPrimeLimitA,
                    uint32_t nPrimeLimitB,
                    uint32_t nBitArray_Size)
{
    uint32_t nThreads = nPrimeLimitB - nPrimeLimitA;
    uint32_t nThreadsPerBlock = 128;
    uint32_t nBlocks = (nThreads + nThreadsPerBlock - 1) / nThreadsPerBlock;

    dim3 block(nThreadsPerBlock);
    dim3 grid(nBlocks);

    switch(nOffsetsA)
    {
        case 1: KERNEL_B_LAUNCH(1); break;
        case 2: KERNEL_B_LAUNCH(2); break;
        case 3: KERNEL_B_LAUNCH(3); break;
        case 4: KERNEL_B_LAUNCH(4); break;
        case 5: KERNEL_B_LAUNCH(5); break;
        case 6: KERNEL_B_LAUNCH(6); break;
        case 7: KERNEL_B_LAUNCH(7); break;
        case 8: KERNEL_B_LAUNCH(8); break;
    }
}


#define KERNEL_C_LAUNCH(X)   primesieve_kernelC<<<grid, block, 0, d_Streams[thr_id][str_id]>>>( \
d_origins[thr_id], \
frameResources[thr_id].d_bit_array_sieve[frame_index], \
nBitArray_Size, \
d_primesInverseInvk[thr_id], \
d_base_remainders[thr_id], \
nPrimeLimitB, \
nPrimeLimit, \
X, \
origin_index )

void kernelC_launch(uint8_t thr_id,
                    uint8_t str_id,
                    uint32_t origin_index,
                    uint8_t frame_index,
                    uint32_t nPrimeLimitB,
                    uint32_t nPrimeLimit,
                    uint32_t nBitArray_Size)
{
    uint32_t nThreads = nPrimeLimit - nPrimeLimitB;
    uint32_t nThreadsPerBlock = 128;
    uint32_t nBlocks = (nThreads + nThreadsPerBlock - 1) / nThreadsPerBlock;

    dim3 block(nThreadsPerBlock);
    dim3 grid(nBlocks);

    switch(nOffsetsA)
    {
        case 8: KERNEL_C_LAUNCH(8); break;
        case 7: KERNEL_C_LAUNCH(7); break;
        case 6: KERNEL_C_LAUNCH(6); break;
        case 5: KERNEL_C_LAUNCH(5); break;
        case 4: KERNEL_C_LAUNCH(4); break;
        case 3: KERNEL_C_LAUNCH(3); break;
        case 2: KERNEL_C_LAUNCH(2); break;
        case 1: KERNEL_C_LAUNCH(1); break;






    }
}

void kernel_clear_launch(uint8_t thr_id, uint8_t str_id,
                         uint8_t curr_sieve, uint32_t nBitArray_Size)
{
    uint32_t sharedSizeBits = 32 * 1024 * 8;
    uint32_t allocSize = ((nBitArray_Size*16 + sharedSizeBits-1) / sharedSizeBits) * sharedSizeBits;

    uint32_t nSieveWords = (allocSize + 31) >> 5;

    dim3 block(64);
    dim3 grid((nSieveWords + block.x - 1) / block.x);

    clearsieve_kernel<<<grid, block, 0, d_Streams[thr_id][str_id]>>>(
    frameResources[thr_id].d_bit_array_sieve[curr_sieve], nSieveWords);
}

extern "C" bool cuda_primesieve(uint8_t thr_id,
                                uint64_t primorial,
                                uint16_t nPrimorialEndPrime,
                                uint16_t nPrimeLimitA,
                                uint32_t nPrimeLimitB,
                                uint32_t nPrimeLimit,
                                uint32_t nBitArray_Size,
                                uint32_t nDifficulty,
                                uint32_t sieve_index,
                                uint32_t test_index,
                                uint32_t nOrigins,
                                uint32_t nMaxCandidates)
{
    /* Get the current working sieve and test indices */
    uint8_t prev_sieve = (sieve_index - 1) % FRAME_COUNT;
    uint8_t curr_sieve = sieve_index % FRAME_COUNT;
    uint8_t curr_test = test_index % FRAME_COUNT;
    uint32_t next_test = (test_index + 1) % FRAME_COUNT;
    uint32_t prev_test = (test_index - 1) % FRAME_COUNT;

    /* Make sure current working sieve is finished */
    if(hipEventQuery(d_Events[thr_id][curr_sieve][EVENT::COMPACT]) == hipErrorNotReady
    || hipEventQuery(d_Events[thr_id][prev_test ][EVENT::FERMAT ]) == hipErrorNotReady)
        return false;
    //CHECK(synchronize_event(thr_id, curr_sieve, EVENT::COMPACT));
    //CHECK(synchronize_event(thr_id, prev_test, EVENT::FERMAT));

    uint8_t nComboThreshold = 8;

    uint32_t origin_index = sieve_index % nOrigins;


    {
        /* Wait for testing and compaction to finish before starting next round. */
        //CHECK(stream_wait_event(thr_id, curr_sieve, STREAM::CLEAR, EVENT::COMPACT));
        //CHECK(stream_wait_event(thr_id, prev_test, STREAM::CLEAR, EVENT::FERMAT));

        /* Clear the current working sieve and signal */
        //kernel_clear_launch(thr_id, STREAM::CLEAR, curr_sieve, nBitArray_Size);


        //CHECK(stream_signal_event(thr_id, curr_sieve, STREAM::CLEAR, EVENT::CLEAR));
    }


    {
        CHECK(stream_wait_event(thr_id, prev_sieve, STREAM::SIEVE_A, EVENT::COMPACT));
        CHECK(stream_wait_event(thr_id, prev_test,  STREAM::SIEVE_A, EVENT::FERMAT));


        /* Single sieve (Launch small sieve, utilizing shared memory and signal) */
        kernelA_launch(thr_id, STREAM::SIEVE_A, origin_index, curr_sieve,
                      nPrimorialEndPrime, nPrimeLimitA, nBitArray_Size);

        CHECK(stream_signal_event(thr_id, curr_sieve, STREAM::SIEVE_A, EVENT::SIEVE_A));
        CHECK(stream_wait_event(thr_id,   curr_sieve, STREAM::SIEVE_B, EVENT::SIEVE_A));

        /* Single sieve (Launch large sieve, utilizing global memory and signal) */
        kernelB_launch(thr_id, STREAM::SIEVE_B, origin_index, curr_sieve,
                      nPrimeLimitA, nPrimeLimitB, nBitArray_Size);

        kernelC_launch(thr_id, STREAM::SIEVE_B, origin_index, curr_sieve,
                      nPrimeLimitB, nPrimeLimit, nBitArray_Size);

        CHECK(stream_signal_event(thr_id, curr_sieve, STREAM::SIEVE_B, EVENT::SIEVE_B));
    }


    {
        CHECK(stream_wait_event(thr_id, curr_sieve, STREAM::SIEVE_A, EVENT::SIEVE_B));


        /* Combo sieve (Launch small sieve, utilizing shared memory and signal) */
        comboA_launch(thr_id, STREAM::SIEVE_A, origin_index, curr_sieve,
                    nPrimorialEndPrime, nPrimeLimitA, nBitArray_Size, nOrigins);

        CHECK(stream_signal_event(thr_id, curr_sieve, STREAM::SIEVE_A, EVENT::SIEVE_A));
        CHECK(stream_wait_event(thr_id,  curr_sieve,  STREAM::SIEVE_B, EVENT::SIEVE_A));

        /* Combo sieve (Launch large sieve, utilizing global memory and signal) */
        comboB_launch(thr_id, STREAM::SIEVE_B, origin_index, curr_sieve,
                      nPrimeLimitA, nPrimeLimitB, nBitArray_Size);

        CHECK(stream_signal_event(thr_id, curr_sieve, STREAM::SIEVE_B, EVENT::SIEVE_B));
    }


    {   /* Launch compaction and signal */
        CHECK(stream_wait_events(thr_id, curr_sieve, STREAM::COMPACT, EVENT::SIEVE_A, EVENT::SIEVE_B));

        kernel_ccompact_launch(thr_id, STREAM::COMPACT, origin_index, nMaxCandidates, curr_sieve, curr_test, next_test, nBitArray_Size, nComboThreshold);

        CHECK(stream_signal_event(thr_id, curr_sieve, STREAM::COMPACT, EVENT::COMPACT));
    }


    debug::log(4, FUNCTION, (uint32_t)thr_id, ": origin index=", sieve_index);

    return true;
}

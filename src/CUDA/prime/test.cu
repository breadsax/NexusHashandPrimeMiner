#include "hip/hip_runtime.h"
/*******************************************************************************************

 Nexus Earth 2018

 [Scale Indefinitely] BlackJack. http://www.opensource.org/licenses/mit-license.php

*******************************************************************************************/
#include <CUDA/include/test.h>
#include <CUDA/include/fermat.cuh>
#include <CUDA/include/util.h>
#include <CUDA/include/frame_resources.h>

#include <CUDA/include/streams_events.h>

#include <CUDA/include/constants.h>

#include <Util/include/debug.h>
#include <Util/include/prime_config.h>

#include <stdio.h>
#include <algorithm>


hipError_t d_result_event_curr[GPU_MAX][FRAME_COUNT];
hipError_t d_result_event_prev[GPU_MAX][FRAME_COUNT];


extern "C" void cuda_set_primorial(uint8_t thr_id, uint64_t nPrimorial)
{
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_primorial), &nPrimorial,
        sizeof(uint64_t), 0, hipMemcpyHostToDevice));
}


extern "C" void cuda_set_BaseOrigin(uint32_t thr_id, uint32_t *limbs)
{
    debug::log(4, FUNCTION, thr_id);

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_zBaseOrigin), limbs,
        WORD_MAX*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
}


extern "C" void cuda_set_quit(uint32_t quit)
{
    debug::log(4, FUNCTION, quit ? "true" : "false");

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_quit), &quit,
                            sizeof(uint32_t), 0, hipMemcpyHostToDevice));

}


__global__ void precompact_test_offsets(uint64_t *in_nonce_offsets,
                                     uint32_t *in_nonce_meta,
                                     uint32_t *in_nonce_count,
                                     uint64_t *out_nonce_offsets,
                                     uint32_t *out_nonce_meta,
                                     uint32_t *out_nonce_count,
                                     uint32_t nMaxCandidates)
{
    /* If the quit flag was set, early return to avoid wasting time. */
    if(c_quit)
        return;

    /* Compute the global index for this nonce offset. */
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < *in_nonce_count)
    {
        uint64_t nonce_offset = in_nonce_offsets[idx];
        uint32_t nonce_meta = in_nonce_meta[idx];

        /* Get the count of tested bits. */
        uint32_t nCount = __popc(nonce_meta & c_bitmaskT);

        /* If the count meets the threshold, add to result buffer. */
        if(nCount)
            add_result(out_nonce_offsets, out_nonce_meta, out_nonce_count,  nonce_offset, nonce_meta, nMaxCandidates);
    }
}


__global__ void compact_test_offsets(uint64_t *in_nonce_offsets,
                                     uint32_t *in_nonce_meta,
                                     uint32_t *in_nonce_count,
                                     uint64_t *out_nonce_offsets,
                                     uint32_t *out_nonce_meta,
                                     uint32_t *out_nonce_count,
                                     uint32_t nThreshold,
                                     uint32_t nMaxCandidates)
{
    /* If the quit flag was set, early return to avoid wasting time. */
    if(c_quit)
        return;

    /* Compute the global index for this nonce offset. */
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < *in_nonce_count)
    {
        uint64_t nonce_offset = in_nonce_offsets[idx];
        uint32_t nonce_meta = in_nonce_meta[idx];

        /* Mask out the tested bits. */
        uint32_t test_combo = nonce_meta & c_bitmaskT;

        /* Reverse tested bits to match sieved bits. */
        nonce_meta = nonce_meta ^ c_bitmaskT;

        /* Get the count of tested bits. */
        uint32_t nCount = __popc(test_combo);

        /* If the count meets the threshold, add to result buffer. */
        if(nCount >= nThreshold)
        {
            //printf("%d: compact_fermat: nonce_meta=%08X, test_combo=%08X, count=%d\n", idx, nonce_meta, test_combo, nCount);

            add_result(out_nonce_offsets, out_nonce_meta, out_nonce_count,
                       nonce_offset, nonce_meta, nMaxCandidates);
        }
    }
}

/* Fermat Test and sort offsets into resulting or working buffers. */
__global__ void fermat_kernel(uint64_t *nonce_offsets,
                              uint32_t *nonce_meta,
                              uint32_t *nonce_count,
                              uint32_t *window_data,
                              uint32_t *g_primes_checked,
                              uint32_t *g_primes_found,
                              uint32_t o)
{
    /* If the quit flag was set, early return to avoid wasting time. */
    if(c_quit)
        return;

    /* Compute the global index for this nonce offset. */
    uint32_t position = blockIdx.x * blockDim.x + threadIdx.x;

    /* Make sure index is not out of bounds. */
    if(position < *nonce_count)
    {
        uint32_t p[WORD_MAX];
        uint32_t test_index = c_iT[o];

        /* Compute the primorial offset from the primorial and
         * offset pattern (i.e (293257 + 510510*n) + [0,4,6,10] ).
         * Add to the first sieving element to compute prime to test. */
        add_ui(p, c_zBaseOrigin, nonce_offsets[position] + (uint64_t)c_offsets[test_index]);

        /* Check if prime passes fermat test base 2. */
        uint32_t prime = fermat_prime(p, &window_data[position * WINDOW_SIZE * WORD_MAX]);

        /* Increment primes found. */
        atomicAdd(&g_primes_found[test_index], prime);

        /* Increment primes checked. */
        atomicAdd(&g_primes_checked[test_index], 1);

        /* Update the nonce combo. */
        atomicOr(&nonce_meta[position], prime << test_index);
    }

}


extern "C" __host__ void cuda_fermat(uint32_t thr_id,
                                     uint32_t sieve_index,
                                     uint32_t test_index,
                                     uint32_t nTestLevels,
                                     uint32_t nMaxCandidates)
{
    uint32_t curr_sieve = sieve_index % FRAME_COUNT;
    uint32_t curr_test = test_index % FRAME_COUNT;


    debug::log(4, FUNCTION, thr_id);

    /* Set the result event switch. */
    d_result_event_curr[thr_id][curr_test] = hipErrorNotReady;
    d_result_event_prev[thr_id][curr_test] = hipErrorNotReady;


    /*Make sure compaction event is finished before testing. */
    CHECK(stream_wait_event(thr_id, curr_sieve, STREAM::FERMAT, EVENT::COMPACT));
    //CHECK(synchronize_event(thr_id, curr_sieve, EVENT::COMPACT));

    /* Reset host-side counts to zero. */
    *frameResources[thr_id].h_result_count[curr_test] = 0;
    for(uint8_t i = 0; i < OFFSETS_MAX; ++i)
    {
        frameResources[thr_id].h_primes_checked[curr_test][i] = 0;
        frameResources[thr_id].h_primes_found[curr_test][i] = 0;
    }

    /* Copy zeroed-out primes checked. */
    CHECK(hipMemcpyAsync(frameResources[thr_id].d_primes_checked[curr_test],
                          frameResources[thr_id].h_primes_checked[curr_test],
                          OFFSETS_MAX * sizeof(uint32_t), hipMemcpyHostToDevice, d_Streams[thr_id][STREAM::FERMAT]));

    /* Copy zeroed-out primes found. */
    CHECK(hipMemcpyAsync(frameResources[thr_id].d_primes_found[curr_test],
                          frameResources[thr_id].h_primes_found[curr_test],
                          OFFSETS_MAX * sizeof(uint32_t), hipMemcpyHostToDevice, d_Streams[thr_id][STREAM::FERMAT]));

    /* Copy zeroed-out result count. */
    CHECK(hipMemcpyAsync(frameResources[thr_id].d_result_count[curr_test],
                          frameResources[thr_id].h_result_count[curr_test],
                          sizeof(uint32_t), hipMemcpyHostToDevice, d_Streams[thr_id][STREAM::FERMAT]));


    uint32_t nThreads = *frameResources[thr_id].h_nonce_count[curr_test];

    /* Make sure there are candidates. */
    if(nThreads == 0)
        return;

    if(nThreads >= nMaxCandidates)
    {
        debug::error(FUNCTION, "Max Candidates limit reached: ", nThreads, "/", nMaxCandidates);
        return;
    }


    debug::log(3, FUNCTION, (uint32_t)thr_id,
        ": nonce_count = ", nThreads,
        " queue filled = ", (nThreads * 100.0) / nMaxCandidates, "%");



    dim3 block(256);
    dim3 grid((nThreads + block.x - 1) / block.x);

    uint32_t sharedSizeBytes = 0;//threadsPerBlock * WORD_MAX * sizeof(uint32_t);

    /* how many pretests to find one prime. */
    uint32_t nPreTest = 4;

    /* Launch the fermat testing kernel (pretest). */
    for(uint32_t o = 0; o < nPreTest; ++o)
    {
        fermat_kernel<<<grid, block, sharedSizeBytes, d_Streams[thr_id][STREAM::FERMAT]>>>(
            frameResources[thr_id].d_pre_nonce_offsets[curr_test],
            frameResources[thr_id].d_pre_nonce_meta[curr_test],
            frameResources[thr_id].d_pre_nonce_count[curr_test],
            frameResources[thr_id].d_window_data[curr_test],
            frameResources[thr_id].d_primes_checked[curr_test],
            frameResources[thr_id].d_primes_found[curr_test],
            o);
    }

    dim3 block2(128);
    dim3 grid2((nThreads + block2.x - 1) / block2.x);

    /* Compact results down into result buffer. */
    precompact_test_offsets<<<grid2, block2, 0, d_Streams[thr_id][STREAM::FERMAT]>>>(
        frameResources[thr_id].d_pre_nonce_offsets[curr_test],
        frameResources[thr_id].d_pre_nonce_meta[curr_test],
        frameResources[thr_id].d_pre_nonce_count[curr_test],
        frameResources[thr_id].d_nonce_offsets[curr_test],
        frameResources[thr_id].d_nonce_meta[curr_test],
        frameResources[thr_id].d_nonce_count[curr_test],
        nMaxCandidates);

    /* Launch the fermat testing kernel. */
    for(uint32_t o = nPreTest; o < vOffsetsT.size(); ++o)
    {
        fermat_kernel<<<grid, block, sharedSizeBytes, d_Streams[thr_id][STREAM::FERMAT]>>>(
            frameResources[thr_id].d_nonce_offsets[curr_test],
            frameResources[thr_id].d_nonce_meta[curr_test],
            frameResources[thr_id].d_nonce_count[curr_test],
            frameResources[thr_id].d_window_data[curr_test],
            frameResources[thr_id].d_primes_checked[curr_test],
            frameResources[thr_id].d_primes_found[curr_test],
            o);
    }


    /* Compact results down into result buffer. */
    compact_test_offsets<<<grid2, block2, 0, d_Streams[thr_id][STREAM::FERMAT]>>>(
        frameResources[thr_id].d_nonce_offsets[curr_test],
        frameResources[thr_id].d_nonce_meta[curr_test],
        frameResources[thr_id].d_nonce_count[curr_test],
        frameResources[thr_id].d_result_offsets[curr_test],
        frameResources[thr_id].d_result_meta[curr_test],
        frameResources[thr_id].d_result_count[curr_test],
        nTestLevels,
        nMaxCandidates);

    /* Copy the result count. */
    CHECK(hipMemcpyAsync(frameResources[thr_id].h_result_count[curr_test],
                          frameResources[thr_id].d_result_count[curr_test],
                          sizeof(uint32_t), hipMemcpyDeviceToHost, d_Streams[thr_id][STREAM::FERMAT]));

    /* Copy the result offsets. */
    CHECK(hipMemcpyAsync(frameResources[thr_id].h_result_offsets[curr_test],
                          frameResources[thr_id].d_result_offsets[curr_test],
                          nMaxCandidates * sizeof(uint64_t), hipMemcpyDeviceToHost, d_Streams[thr_id][STREAM::FERMAT]));

    /* copy the result meta. */
    CHECK(hipMemcpyAsync(frameResources[thr_id].h_result_meta[curr_test],
                          frameResources[thr_id].d_result_meta[curr_test],
                          nMaxCandidates * sizeof(uint32_t), hipMemcpyDeviceToHost, d_Streams[thr_id][STREAM::FERMAT]));

    /* Copy the amount of primes checked. */
    CHECK(hipMemcpyAsync(frameResources[thr_id].h_primes_checked[curr_test],
                          frameResources[thr_id].d_primes_checked[curr_test],
                          OFFSETS_MAX * sizeof(uint32_t), hipMemcpyDeviceToHost, d_Streams[thr_id][STREAM::FERMAT]));

    /* Copy the amount of primes found. */
    CHECK(hipMemcpyAsync(frameResources[thr_id].h_primes_found[curr_test],
                          frameResources[thr_id].d_primes_found[curr_test],
                          OFFSETS_MAX * sizeof(uint32_t), hipMemcpyDeviceToHost, d_Streams[thr_id][STREAM::FERMAT]));

    /* Signal the Fermat event. */
    CHECK(stream_signal_event(thr_id, curr_test, STREAM::FERMAT, EVENT::FERMAT));
}


extern "C" void cuda_results(uint32_t thr_id,
                             uint32_t test_index,
                             uint64_t *result_offsets,
                             uint32_t *result_meta,
                             uint32_t *result_count,
                             uint32_t *primes_checked,
                             uint32_t *primes_found,
                             bool fSynchronize)
{
    /* Clear the stats. */
    *result_count = 0;
    for(uint16_t i = 0; i < OFFSETS_MAX; ++i)
    {
        primes_checked[i] = 0;
        primes_found[i] = 0;
    }

    uint32_t curr_test = test_index % FRAME_COUNT;

    if(fSynchronize)
        synchronize_event(thr_id, curr_test, EVENT::FERMAT);

    d_result_event_prev[thr_id][curr_test] = d_result_event_curr[thr_id][curr_test];
    d_result_event_curr[thr_id][curr_test] = hipEventQuery(d_Events[thr_id][curr_test][EVENT::FERMAT]);

    if(d_result_event_curr[thr_id][curr_test] == hipSuccess &&
       d_result_event_prev[thr_id][curr_test] == hipErrorNotReady)
    {
        /* Reset event to trigger one way switch. */
        d_result_event_prev[thr_id][curr_test] = hipSuccess;

        *result_count   = *frameResources[thr_id].h_result_count[curr_test];


        /* Update the primes checked/found for each offset from GPU. */
        for(uint32_t i = 0; i < OFFSETS_MAX; ++i)
        {
            primes_checked[i] =  frameResources[thr_id].h_primes_checked[curr_test][i];
            primes_found[i] =  frameResources[thr_id].h_primes_found[curr_test][i];
        }

        if(*result_count == 0)
            return;

        uint64_t *pOffsets = frameResources[thr_id].h_result_offsets[curr_test];
        uint32_t *pMeta  =   frameResources[thr_id].h_result_meta[curr_test];

        std::copy(pOffsets, pOffsets + (*result_count), result_offsets);
        std::copy(pMeta,    pMeta    + (*result_count), result_meta);

        debug::log(4, FUNCTION, thr_id, "    ", *result_count, " results");
    }
}

extern "C" void cuda_init_counts(uint32_t thr_id)
{
    debug::log(4, FUNCTION, thr_id);

    CHECK(hipDeviceSynchronize());

    for(int i = 0; i < FRAME_COUNT; ++i)
    {
        *frameResources[thr_id].h_nonce_count[i] = 0;

        CHECK(hipMemcpy(frameResources[thr_id].d_nonce_count[i],
                         frameResources[thr_id].h_nonce_count[i],
                         sizeof(uint32_t),
                         hipMemcpyHostToDevice));

        CHECK(hipMemcpy(frameResources[thr_id].d_pre_nonce_count[i],
                         frameResources[thr_id].h_nonce_count[i],
                         sizeof(uint32_t),
                         hipMemcpyHostToDevice));
    }
}
